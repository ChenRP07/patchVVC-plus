#include "hip/hip_runtime.h"
/* Copyright Notice.
 *
 * Please read the LICENSE file in the project root directory for details
 * of the open source licenses referenced by this source code.
 *
 * Copyright: @SDUCS_IIC. All Right Reserved.
 *
 * Author        : Lixin
 * Description   :
 * Create Time   : 2023/05/02 16:54
 * Last Modified : 2023/05/02 16:54
 *
 */
#include "cuda/octree.cuh"
#include "cuda/entropy_codec.cuh"
#include <stdio.h>

namespace vvc {
namespace client{
namespace octree {
    __device__ void InvertHaarTransform(int _w0, int _w1, const common::ColorYUV& _g0, const common::ColorYUV& _g1, common::ColorYUV& _res0, common::ColorYUV& _res1) {
		common::ColorYUV G{}, H{};
		if (_w0 == 0 && _w1 == 0) {
            _res0 = G;
            _res1 = H;
			return ;
		}

		/*
		 *        [ √w0 -√w1]
		 *        [ √w1 √w0 ]
		 * T^-1 = -----------
		 *        √(w0 + w1)
		 *
		 * */
		float base    = sqrtf(static_cast<float>(_w0 + _w1));
		float T[2][2] = {{sqrtf(static_cast<float>(_w0)) / base, -sqrt(static_cast<float>(_w1)) / base},
		                 {sqrtf(static_cast<float>(_w1)) / base, sqrt(static_cast<float>(_w0)) / base}};

		G = _g0 * T[0][0] + _g1 * T[0][1];
		H = _g0 * T[1][0] + _g1 * T[1][1];
        _res0 = G;
        _res1 = H;
		return ;
	}

    __device__ void LoadTreeCore(common::PointXYZ& _center, common::PointXYZ& _range, int& _height, uint8_t (&_p)[25]) {
		float data[6] = {};

		for (int i = 0; i < 6; ++i) {
			int d{0};
			for (int j = 0; j < 4; ++j) {
				d <<= 8;
				d |= _p[i * 4 + j];
			}
			float* t = reinterpret_cast<float*>(&d);
			data[i]  = *t;
		}
		_center.x = data[0], _center.y = data[1], _center.z = data[2];
		_range.x = data[3], _range.y = data[4], _range.z = data[5];
        uint8_t temp = _p[24];
		_height = static_cast<int>(temp);
	}

    __device__ common::PointXYZ SubSpaceCenter(common::PointXYZ _center, common::PointXYZ _range, int _pos) {
        switch (_pos) {
            case 0: return common::PointXYZ(_center.x + _range.x / 2.0f, _center.y + _range.y / 2.0f, _center.z + _range.z / 2.0f);
            case 1: return common::PointXYZ(_center.x + _range.x / 2.0f, _center.y + _range.y / 2.0f, _center.z - _range.z / 2.0f);
            case 2: return common::PointXYZ(_center.x + _range.x / 2.0f, _center.y - _range.y / 2.0f, _center.z + _range.z / 2.0f);
            case 3: return common::PointXYZ(_center.x + _range.x / 2.0f, _center.y - _range.y / 2.0f, _center.z - _range.z / 2.0f);
            case 4: return common::PointXYZ(_center.x - _range.x / 2.0f, _center.y + _range.y / 2.0f, _center.z + _range.z / 2.0f);
            case 5: return common::PointXYZ(_center.x - _range.x / 2.0f, _center.y + _range.y / 2.0f, _center.z - _range.z / 2.0f);
            case 6: return common::PointXYZ(_center.x - _range.x / 2.0f, _center.y - _range.y / 2.0f, _center.z + _range.z / 2.0f);
            case 7: return common::PointXYZ(_center.x - _range.x / 2.0f, _center.y - _range.y / 2.0f, _center.z - _range.z / 2.0f);
            default: printf("SubSpaceCenter ERROR\n"); return common::PointXYZ();
        }
	}

    __device__ void OctreeNode_t::InvertHierarchicalTransform() {
		common::ColorYUV H[8];
		for (int i = 0; i < 8; ++i) {
			H[i] = this->raht[i];
		}
		/* g_DC */
		this->raht[1] = H[0];
		/* Invert X/Y/Z merge */
		for (int i = 1; i < 8; ++i) {
			// std::pair<common::ColorYUV, common::ColorYUV> g(this->raht[i], H[i]), res;
			// std::pair<int, int>                           w(this->weight[NodeWeight[i][0]], this->weight[NodeWeight[i][1]]);
			/* Inver haar wavelet transform */
			InvertHaarTransform(this->weight[NodeWeight[i][0]], this->weight[NodeWeight[i][1]], this->raht[i], H[i], this->raht[NodeWeight[i][0]], this->raht[NodeWeight[i][1]]);
		}
	}

    __device__ void InvertRAHTOctree::SetSlice(const common::Slice_t& _slice) {
        this->slice_ = _slice;
        /* Optional Zstd decoding */
        this->node_values_ = this->slice_.geometry;
        auto temp_color = this->slice_.color;
        /* RLGR decoding */
        common::RLGRDecoder rlgr_dec;
        rlgr_dec.Decode(temp_color, this->slice_.color_size, 3 * this->slice_.size);
        auto rlgr_res       = rlgr_dec.GetResult();
        free(this->coefficients_);
        this->coefficients_ = (common::ColorYUV*)malloc(sizeof(common::ColorYUV) * this->slice_.size);
        /* Reconstruct coefficients */
        for (int i = 0; i < this->slice_.size; ++i) {
            this->coefficients_[i].y = static_cast<float>(rlgr_res[i] * this->slice_.qp);
            this->coefficients_[i].u = static_cast<float>(rlgr_res[i + this->slice_.size] * this->slice_.qp);
            this->coefficients_[i].v = static_cast<float>(rlgr_res[i + 2 * this->slice_.size] * this->slice_.qp);
        }
        free(rlgr_res);
        /* If intra slice, clear tree and related container */
        if (!common::CheckSliceType(this->slice_.type, common::PVVC_SLICE_TYPE_PREDICT)) {
            for (int height = 0; height < this->tree_height_; ++height) {
                delete []this->tree_[height].nodes;
            }
            free(this->tree_);
            free(this->source_cloud_);
            this->source_cloud_index_ = 0;
            free(this->reference_colors_);
            free(this->source_colors_);

            this->source_cloud_ = (common::PointXYZ*)malloc(sizeof(common::PointXYZ) * this->slice_.size);
            this->reference_colors_ = (common::ColorYUV*)malloc(sizeof(common::ColorYUV) * this->slice_.size);
            this->source_colors_ = (common::ColorYUV*)malloc(sizeof(common::ColorYUV) * this->slice_.size);
            this->MakeTree();
        }
        this->InvertRAHT();
	} 

    __device__ void InvertRAHTOctree::GetPatch()  {
        /* Generate Patch and copy data */
        printf("this->slice_.timestamp = %d\n",this->slice_.timestamp);
        printf("this->slice_.index = %d\n",this->slice_.index);
        for(uint32_t i=0; i<4; i++){
            for (uint32_t j=0; j<4; j++){
                printf("%.2f ", this->slice_.mv(i,j));
            }
            printf("\n");
        }
        for(int i=0; i<source_cloud_index_; i++){
            float r = ((this->source_colors_[i].y + 1.4020f * (this->source_colors_[i].v - 128.0f)));
			float g = ((this->source_colors_[i].y - 0.3441f * (this->source_colors_[i].u - 128.0f) - 0.7141f * (this->source_colors_[i].v - 128.0f)));
			float b = ((this->source_colors_[i].y + 1.7720f * (this->source_colors_[i].u - 128.0f)));
            printf("(%.3f,%.3f,%.3f - %.0f,%.0f,%.0f)\n", this->source_cloud_[i].x, this->source_cloud_[i].y, this->source_cloud_[i].z, r, g, b);
        }
	}

    __device__ void InvertRAHTOctree::MakeTree() {
        /* Load center, range and height from geometry */
        uint8_t tree_attr[25]{};

        int node_values_index = 0;
        for (; node_values_index < 25; node_values_index++) {
            tree_attr[node_values_index] = node_values_[node_values_index];
        }

        LoadTreeCore(this->tree_center_, this->tree_range_, this->tree_height_, tree_attr);
        
        this->tree_ = (OctreeLayer_t *)malloc(sizeof(OctreeLayer_t) * this->tree_height_);
        int curr_layer_node_count = 1;
        common::ColorYUV zero{};
        /* Assign value for each branch node */
        for (int idx = 0; idx < this->tree_height_ - 1; ++idx) {
            /* Count how many nodes next layer has */
            int next_layer_node_count = 0;
            /* Assign nodes in current layer */
            this->tree_[idx].nodes = new OctreeNode_t[curr_layer_node_count];
            this->tree_[idx].length = curr_layer_node_count;
            for (int cnt = 0; cnt < curr_layer_node_count; ++cnt) {
                /* Set value */
                this->tree_[idx].nodes[cnt].value = node_values_[node_values_index];
                /* Count 1-bits of this node */
                for ( int i=0; i<8; i++){
                    if (((node_values_[node_values_index]) & NodeValue[i]) == 0) {
                        this->tree_[idx].nodes[cnt].index[i] = -1;
                    }
                    else{
                        this->tree_[idx].nodes[cnt].index[i] = next_layer_node_count;
                        ++next_layer_node_count;
                    }
                }
                node_values_index++;
            }
            curr_layer_node_count = next_layer_node_count;
        }

        /* Malloc space for last layer */
        this->tree_[this->tree_height_ - 1].nodes = (OctreeNode_t *)malloc(sizeof(OctreeNode_t) * curr_layer_node_count);
        this->tree_[this->tree_height_ - 1].length = curr_layer_node_count;
        
        /* Update center and range for each node */
        this->tree_[0].nodes[0].center = this->tree_center_;
        this->tree_[0].nodes[0].range  = this->tree_range_;
        for (int layer = 0; layer < this->tree_height_ - 1; ++layer) {
            for (int idx = 0; idx < this->tree_[layer].length; ++idx) { 
                auto& node = this->tree_[layer].nodes[idx];
                common::PointXYZ subrange(node.range.x / 2.0f, node.range.y / 2.0f, node.range.z / 2.0f);
                for (int i = 0; i < 8; ++i) {
                    if (node.index[i] != -1) {
                        this->tree_[layer + 1].nodes[node.index[i]].center = SubSpaceCenter(node.center, subrange, i);
                        this->tree_[layer + 1].nodes[node.index[i]].range = subrange;
                    }     
                }
            }
        }
        
        /* Collect last layer centers and update weight for last layer */
        for (int idx = 0; idx < this->tree_[this->tree_height_ - 1].length; ++idx) {
            auto& node = this->tree_[this->tree_height_ - 1].nodes[idx];
            node.value = 0xff;
            node.weight[1] = 1;
            this->source_cloud_[this->source_cloud_index_] = node.center;
            node.index[0] = this->source_cloud_index_;
            this->source_cloud_index_++;
        }

        /* Reversely update weight for each node */
        for (int layer = this->tree_height_ - 2; layer >= 0; --layer) {
            for (int idx = 0; idx < this->tree_[layer].length; ++idx) {
                auto& node = this->tree_[layer].nodes[idx];
                for (int i = 0; i < 8; ++i) {
                    if (node.index[i] != -1) {
                        node.weight[i + 8] = this->tree_[layer + 1].nodes[node.index[i]].weight[1];
                    }
                }
                for (int i = 7; i > 0; --i) {
                    node.weight[i] = node.weight[NodeWeight[i][0]] + node.weight[NodeWeight[i][1]];
                }
            }
        }
        /* Update weight and add point into cloud */
	}

    __device__ void InvertRAHTOctree::InvertRAHT() {
        
        /* coefficients_ and source_cloud_ have the same size */
        int coefficients_index = this->source_cloud_index_ - 1;

        /* Set g_DC */
        this->tree_[0].nodes[0].raht[0] = this->coefficients_[coefficients_index];
        coefficients_index--;

        for (int i = 0; i < this->tree_height_ - 1; ++i) {
            for(int j = 0; j < this->tree_[i].length; j++){
                auto &node = this->tree_[i].nodes[j];
                /* Set h_ACs */
                for (int idx = 1; idx < 8; ++idx) {
                    if (node.weight[NodeWeight[idx][0]] != 0 && node.weight[NodeWeight[idx][1]] != 0) {
                        node.raht[idx] = this->coefficients_[coefficients_index];
                        coefficients_index --;
                    }
                }
                /* Compute g_DC */
                node.InvertHierarchicalTransform();

                /* Update g_DC for each subnode */
                for (int idx = 0; idx < 8; ++idx) {
                    if (node.index[idx] != -1) {
                        this->tree_[i + 1].nodes[node.index[idx]].raht[0] = node.raht[idx + 8];
                    }
                }
            }
        }

        /* Collect colors */
        OctreeLayer_t &last_layer = this->tree_[tree_height_ - 1];
        for (int i = 0; i < last_layer.length; i++){
            this->source_colors_[i] = last_layer.nodes[i].raht[0];
        }

        if (common::CheckSliceType(this->slice_.type, common::PVVC_SLICE_TYPE_PREDICT)) {
            for (int i = 0; i < this->source_cloud_index_; ++i) {
                this->source_colors_[i] += this->reference_colors_[i];
            }
        }
        else {
            for(int i=0; i<source_cloud_index_; i++){
                this->reference_colors_[i] = this->source_colors_[i];
            }
        }
    }

}  // namespace octree
}
}  // namespace vvc
