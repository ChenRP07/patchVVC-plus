#include "hip/hip_runtime.h"
/*
 * @Author: lixin
 * @Date: 2023-05-16 11:47:17
 * @LastEditTime: 2023-05-18 16:10:57
 * @Description: 
 * Copyright (c) @lixin, All Rights Reserved.
 */
#include ""
#include <hip/hip_runtime_api.h>
#include "cuda/octree.cuh"

struct Points
{
	float x, y, z;
	float r, g, b;
};

// 起始地址 + Slice的属性 + I帧的数组

/**
 * @description: 函数注释配置模板
 * @param {Points*} cudaData		该帧在 VBO 的起始地址
 * @param {int*} inner_offset		该 Patch 在 帧 内的起始地址
 * @param {int} timestamp			帧序号
 * @param {int*} index				Patch 序号
 * @param {uint8_t*} type			Patch 的解码类型
 * @param {float**} mv				每个 Patch 的 MotionVector
 * @param {uint32_t*} size			每个 Patch 解压后的 点数
 * @param {uint8_t*} qp				
 * @param {uint8_t**} geometry
 * @param {uint32_t*} geometry_size
 * @param {uint8_t**} color
 * @param {uint32_t*} color_size
 * @param {InvertRAHTOctree*} invertRAHTOctree_gpu
 * @return {*}
 */
__global__ void processCUDA(Points* cudaData, int timestamp, int* inner_offset, int* index, uint8_t* type, float** mv, uint32_t* size, uint8_t* qp, uint8_t** geometry, uint32_t* geometry_size, uint8_t** color, uint32_t* color_size, vvc::client::octree::InvertRAHTOctree* invertRAHTOctree_gpu, int patch_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < patch_size){
		vvc::client::common::Slice_t slice(timestamp, index[idx], type[idx], mv[idx], size[idx], qp[idx], geometry[idx], geometry_size[idx], color[idx], color_size[idx]);
		invertRAHTOctree_gpu[index[idx]].SetSlice(slice);
		// 找到帧内偏移
		int offset = inner_offset[idx];
		for(int i=0; i<size[idx]; i++){
			cudaData[offset + i].x = invertRAHTOctree_gpu[index[idx]].source_cloud_[i].x * mv[idx][0] + invertRAHTOctree_gpu[index[idx]].source_cloud_[i].y * mv[idx][1] + invertRAHTOctree_gpu[index[idx]].source_cloud_[i].z * mv[idx][2] + mv[idx][3];
			cudaData[offset + i].y = invertRAHTOctree_gpu[index[idx]].source_cloud_[i].x * mv[idx][4] + invertRAHTOctree_gpu[index[idx]].source_cloud_[i].y * mv[idx][5] + invertRAHTOctree_gpu[index[idx]].source_cloud_[i].z * mv[idx][6] + mv[idx][7];
			cudaData[offset + i].z = invertRAHTOctree_gpu[index[idx]].source_cloud_[i].x * mv[idx][8] + invertRAHTOctree_gpu[index[idx]].source_cloud_[i].y * mv[idx][9] + invertRAHTOctree_gpu[index[idx]].source_cloud_[i].z * mv[idx][10] + mv[idx][11];
			cudaData[offset + i].r = ((invertRAHTOctree_gpu[index[idx]].source_colors_[i].y + 1.4020f * (invertRAHTOctree_gpu[index[idx]].source_colors_[i].v - 128.0f))) / 255;
			cudaData[offset + i].g = ((invertRAHTOctree_gpu[index[idx]].source_colors_[i].y - 0.3441f * (invertRAHTOctree_gpu[index[idx]].source_colors_[i].u - 128.0f) - 0.7141f * (invertRAHTOctree_gpu[index[idx]].source_colors_[i].v - 128.0f))) / 255;
			cudaData[offset + i].b = ((invertRAHTOctree_gpu[index[idx]].source_colors_[i].y + 1.7720f * (invertRAHTOctree_gpu[index[idx]].source_colors_[i].u - 128.0f))) / 255;
		}
		// if(idx == 1){
		// 	// 	invertRAHTOctree_gpu[index[1]].GetPatch();
		// 	for(int i=0; i<size[idx]; i++){
		// 		cudaData[offset + i].x += 100;
		// 	}
		// }
	}
}

extern "C" void launch_cudaProcess(int grid, int block, Points * cudaData, int timestamp, int* inner_offset, int* index, uint8_t* type, float** mv, uint32_t* size, uint8_t* qp, uint8_t** geometry, uint32_t* geometry_size, uint8_t** color, uint32_t* color_size, vvc::client::octree::InvertRAHTOctree* invertRAHTOctree_gpu, int patch_size){
	processCUDA <<<grid, block >>> (cudaData, timestamp, inner_offset, index, type, mv, size, qp, geometry, geometry_size, color, color_size, invertRAHTOctree_gpu, patch_size);
}
