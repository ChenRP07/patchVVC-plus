#include "hip/hip_runtime.h"
/*
 * @Author: lixin
 * @Date: 2023-05-16 11:47:17
 * @LastEditTime: 2023-05-17 16:11:13
 * @Description: 
 * Copyright (c) @lixin, All Rights Reserved.
 */
#include ""
#include <hip/hip_runtime_api.h>
#include "cuda/octree.cuh"

struct Points
{
	float x, y, z;
	float r, g, b;
};

// 起始地址 + Slice的属性 + I帧的数组

/**
 * @description: 函数注释配置模板
 * @param {Points*} cudaData		该帧在 VBO 的起始地址
 * @param {int*} inner_offset		该 Patch 在 帧 内的起始地址
 * @param {int} timestamp			帧序号
 * @param {int*} index				Patch 序号
 * @param {uint8_t*} type			Patch 的解码类型
 * @param {float**} mv				每个 Patch 的 MotionVector
 * @param {uint32_t*} size			每个 Patch 解压后的 点数
 * @param {uint8_t*} qp				
 * @param {uint8_t**} geometry
 * @param {uint32_t*} geometry_size
 * @param {uint8_t**} color
 * @param {uint32_t*} color_size
 * @param {InvertRAHTOctree*} invertRAHTOctree_gpu
 * @return {*}
 */
__global__ void processCUDA(Points* cudaData, int timestamp, int* inner_offset, int* index, uint8_t* type, float** mv, uint32_t* size, uint8_t* qp, uint8_t** geometry, uint32_t* geometry_size, uint8_t** color, uint32_t* color_size, vvc::client::octree::InvertRAHTOctree* invertRAHTOctree_gpu, int patch_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < patch_size){
		// printf("hello1\n");

		// printf("timestamp = %d\n", timestamp);
		// printf("index = %d\n", index[idx]);
		// printf("type = %02x\n", type[idx]);
		// printf("size = %d\n", size[idx]);
		// printf("qp = %d\n", qp[idx]);
		// printf("geometry_size = %d\n", geometry_size[idx]);
		// printf("color_size = %d\n", color_size[idx]);

		// for(int i=0; i<16; i++){
		// 	printf("%.2f ", mv[idx][i]);
		// }
		// printf("\n");

		vvc::client::common::Slice_t slice(timestamp, index[idx], type[idx], mv[idx], size[idx], qp[idx], geometry[idx], geometry_size[idx], color[idx], color_size[idx]);
		// printf("hello2\n");
		invertRAHTOctree_gpu[index[idx]].SetSlice(slice);
		// printf("hello3\n");
		// 找到帧内偏移
		int offset = inner_offset[idx];
		for(int i=0; i<size[idx]; i++){
			cudaData[offset + i].x = invertRAHTOctree_gpu[index[idx]].source_cloud_[i].x;
			cudaData[offset + i].y = invertRAHTOctree_gpu[index[idx]].source_cloud_[i].y;
			cudaData[offset + i].z = invertRAHTOctree_gpu[index[idx]].source_cloud_[i].z;
			cudaData[offset + i].r = ((invertRAHTOctree_gpu[index[idx]].source_colors_[i].y + 1.4020f * (invertRAHTOctree_gpu[index[idx]].source_colors_[i].v - 128.0f))) / 255;
			cudaData[offset + i].g = ((invertRAHTOctree_gpu[index[idx]].source_colors_[i].y - 0.3441f * (invertRAHTOctree_gpu[index[idx]].source_colors_[i].u - 128.0f) - 0.7141f * (invertRAHTOctree_gpu[index[idx]].source_colors_[i].v - 128.0f))) / 255;
			cudaData[offset + i].b = ((invertRAHTOctree_gpu[index[idx]].source_colors_[i].y + 1.7720f * (invertRAHTOctree_gpu[index[idx]].source_colors_[i].u - 128.0f))) / 255;
		}

		// if(idx == 1){
		// 	printf("offset = %d\n",offset);
		// 	invertRAHTOctree_gpu[index[1]].GetPatch();
		// 	for(int i=0; i<size[idx]; i++){
		// 		cudaData[offset + i].x += 100;
		// 	}
		// }
	}
}

extern "C" void launch_cudaProcess(int grid, int block, Points * cudaData, int timestamp, int* inner_offset, int* index, uint8_t* type, float** mv, uint32_t* size, uint8_t* qp, uint8_t** geometry, uint32_t* geometry_size, uint8_t** color, uint32_t* color_size, vvc::client::octree::InvertRAHTOctree* invertRAHTOctree_gpu, int patch_size){
	processCUDA <<<grid, block >>> (cudaData, timestamp, inner_offset, index, type, mv, size, qp, geometry, geometry_size, color, color_size, invertRAHTOctree_gpu, patch_size);
}
